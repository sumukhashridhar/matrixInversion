#include "luBatchedInplace.cuh"

int main() {
    int matrixSize, numMatrices, numThreads;

    std::cout << "Enter matrix size: ";
    std::cin >> matrixSize;
    std::cout << "Enter number of matrices: ";
    std::cin >> numMatrices;
    std::cout << "Enter number of threads in a block: ";
    std::cin >> numThreads;
    // numThreads = 32;
    int threadsPerMatrix = matrixSize;
    int matricesPerBlock = numThreads / threadsPerMatrix;
    int numBlocks = numMatrices / matricesPerBlock + (numMatrices % matricesPerBlock == 0 ? 0 : 1);

    int numElements = matrixSize * matrixSize * numMatrices;

    std::vector<FpType> A(numElements);
    std::vector<FpType> A_inv(numElements);

    FpType* d_A;
    CUDA_CHECK(hipMallocManaged(&d_A, numElements * sizeof(FpType)));

    // Read the file once into a single matrix-sized buffer
    std::ifstream file("mtrand32_new1.txt");
    std::vector<FpType> templateMatrix(matrixSize * matrixSize);
    for (int i = 0; i < matrixSize * matrixSize; ++i) {
        file >> templateMatrix[i];
    }
    file.close();

    #pragma acc parallel loop
    for (int k = 0; k < numMatrices; ++k) {
        int offset = k * matrixSize * matrixSize;
        
        #pragma acc loop vector
        for (int i = 0; i < matrixSize; ++i) {
            for (int j = 0; j < matrixSize; ++j) {
                int templateIndex = (i * matrixSize) + j;
                A[(i * matrixSize) + offset + j] = templateMatrix[templateIndex];
                A_inv[(i * matrixSize) + offset + j] = static_cast<FpType>(0.0);
            }
        }
    }

    CUDA_CHECK(hipMemcpy(d_A, A.data(), numElements * sizeof(FpType), hipMemcpyHostToDevice));
    std::cout << "Data copied to device." << '\n';

    int shMemSize = matricesPerBlock * matrixSize * matrixSize * sizeof(FpType);
    CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(batched_lu_subwarp<FpType>), hipFuncAttributeMaxDynamicSharedMemorySize, shMemSize));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    batched_lu_subwarp<FpType><<<numBlocks, numThreads, shMemSize>>>(d_A, matrixSize, numMatrices, threadsPerMatrix, matricesPerBlock);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Kernel execution time: " << milliseconds << " milliseconds\n";

    CUDA_CHECK(hipMemcpy(A_inv.data(), d_A, numElements * sizeof(FpType), hipMemcpyDeviceToHost));
    std::cout << "Data copied back to host." << '\n';

    // print A_inv
    // printMatrices(A_inv, matrixSize, numMatrices);

    auto startT = std::chrono::high_resolution_clock::now();
    verifyInv(A, A_inv, matrixSize, numMatrices);
    auto endT = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = endT - startT;
    std::cout << "Time taken to verify inverse: " << elapsed.count() << " seconds\n";
    
    hipFree(d_A);

    return 0;
}