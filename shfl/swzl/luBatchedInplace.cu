#include "luBatchedInplace.cuh"

int main() {
    constexpr int matrixSize=MATRIXSIZE;
    constexpr int numMatrices=NUMMATRICES;
    constexpr int numThreads=NUMTHREADS;

    const int threadsPerMatrix = matrixSize;
    const int matricesPerBlock = numThreads / threadsPerMatrix;
    const int numBlocks = numMatrices / matricesPerBlock;

    std::cout << "Matrix size: " << matrixSize << '\n';
    std::cout << "Number of matrices: " << numMatrices << '\n'; 
    std::cout << "Number of threads per block: " << numThreads << '\n';
    std::cout << "Threads per matrix: " << threadsPerMatrix << '\n';
    std::cout << "Matrices per block: " << matricesPerBlock << '\n';
    std::cout << "Number of blocks: " << numBlocks << '\n';

    // FpType inputMatrix[] = {2, 3, 4, 5};//, 10, 4, 2, 4, 2};
     FpType inputMatrix[] = {4, 11, 3, 4, 10, 4, 2, 4, 2};
    // FpType inputMatrix[] = {2, 7, 1, 5, 3, -2, 0, 1, 1, 5, 3, 4, 7, 3, 2, 8};
    // make input matrix as Identity matrix
    // FpType inputMatrix[] = {10, 0, 0, 0, 0, 10, 0, 0, 0, 0, 10, 0, 0, 0, 0, 10};

    int numElements = matrixSize * matrixSize * numMatrices;

    std::vector<FpType> A(numElements);
    std::vector<FpType> A_inv(numElements);

    std::cout << "Reading data from file." << '\n';

        #pragma acc parallel loop
        for (int k = 0; k < numMatrices; ++k) {
            int offset = k * matrixSize * matrixSize;
            // std::ifstream file("main_mtrx.txt");
            std::ifstream file("matrix.txt");
            
            #pragma acc loop vector
            for (int i = 0; i < matrixSize; ++i) {
                for (int j = 0; j < matrixSize; ++j) {
                    file >> A[(i * matrixSize) + offset + j];
                    A_inv[(i * matrixSize) + offset + j] = static_cast<FpType>(0.0);
                }
            }
            file.close();
        }

    std::cout << "Data read from file." << '\n';

    FpType* d_A;
    CUDA_CHECK(hipMalloc(&d_A, numElements * sizeof(FpType)));
    CUDA_CHECK(hipMemcpy(d_A, A.data(), numElements * sizeof(FpType), hipMemcpyHostToDevice));
    std::cout << "Data copied to device." << '\n';

    int shMemSize = matricesPerBlock * matrixSize * matrixSize * sizeof(FpType);
    CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(batched_lu_subwarp<FpType), matrixSize, threadsPerMatrix, matricesPerBlock, numMatrices>, hipFuncAttributeMaxDynamicSharedMemorySize, shMemSize));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    batched_lu_subwarp<FpType, matrixSize, threadsPerMatrix, matricesPerBlock, numMatrices><<<numBlocks, numThreads, shMemSize>>>(d_A);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Kernel execution time: " << milliseconds << " milliseconds\n";

    CUDA_CHECK(hipMemcpy(A_inv.data(), d_A, numElements * sizeof(FpType), hipMemcpyDeviceToHost));
    std::cout << "Data copied back to host." << '\n';

    // print A_inv
    // printf("\n");
    // printMatrices(A, matrixSize, numMatrices);
    // printf("\n");
    // printMatrices(A_inv, matrixSize, numMatrices);

    // write A_inv to file
    // auto filename = "8thread32_mtrx.txt";
    // writeToFile(A_inv, filename, matrixSize, numMatrices);

    auto startT = std::chrono::high_resolution_clock::now();
    verifyInv(A, A_inv, matrixSize, numMatrices);
    // verifyLU(A, A_inv, matrixSize, numMatrices);
    auto endT = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = endT - startT;
    std::cout << "Time taken to verify inverse: " << elapsed.count() << " seconds\n";

    hipFree(d_A);

    return 0;
}