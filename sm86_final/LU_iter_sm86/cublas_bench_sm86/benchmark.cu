#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <chrono>

using FpType = float;

#define CUDA_CHECK(call) { hipError_t err = call; if (err != hipSuccess) { printf("CUDA error: %s (%s:%d)\n", hipGetErrorString(err), __FILE__, __LINE__); exit(1); } }
#define CUBLAS_CHECK(call) { hipblasStatus_t status = call; if (status != HIPBLAS_STATUS_SUCCESS) { printf("cuBLAS error: %d (%s:%d)\n", status, __FILE__, __LINE__); exit(1); } }

int main() {
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    constexpr int BATCH_SIZE = INP_BATCH_SIZE;
    constexpr int N = INP_MATRIX_SIZE;

    std::cout << "Matrix size: " << N << std::endl;
    std::cout << "Batch size: " << BATCH_SIZE << std::endl;

    std::vector<FpType*> h_A(BATCH_SIZE);
    std::vector<FpType*> h_L(BATCH_SIZE);
    std::vector<FpType*> h_U(BATCH_SIZE);

    // Read template matrix
    std::vector<FpType> templateMatrix(N * N);
    {
        std::ifstream file("mtrand32_new1.txt");
        if (!file.is_open()) {
            std::cerr << "Failed to open the file" << std::endl;
            return 1;
        }
        for (int i = 0; i < N * N; ++i) {
            file >> templateMatrix[i];
        }
    }

    // Initialize host matrices with pinned memory
    #pragma omp parallel for
    for (int i = 0; i < BATCH_SIZE; ++i) {
        CUDA_CHECK(hipHostAlloc(&h_A[i], N * N * sizeof(FpType), hipHostMallocDefault));
        CUDA_CHECK(hipHostAlloc(&h_L[i], N * N * sizeof(FpType), hipHostMallocDefault));
        CUDA_CHECK(hipHostAlloc(&h_U[i], N * N * sizeof(FpType), hipHostMallocDefault));
        
        #pragma omp simd
        for (int j = 0; j < N * N; ++j) {
            h_A[i][j] = templateMatrix[j];
            h_L[i][j] = 0.0f;
            h_U[i][j] = 0.0f;
        }
    }

    // Setup CUDA streams
    constexpr int NUM_STREAMS = 4;
    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; ++i) {
        CUDA_CHECK(hipStreamCreate(&streams[i]));
    }

    // Allocate device memory
    FpType **d_A_array;
    int *d_pivots, *d_info;
    CUDA_CHECK(hipMalloc(&d_A_array, BATCH_SIZE * sizeof(FpType*)));
    CUDA_CHECK(hipMalloc(&d_pivots, BATCH_SIZE * N * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_info, BATCH_SIZE * sizeof(int)));

    std::vector<FpType*> d_A(BATCH_SIZE);
    
    // Transfer data using streams
    for (int i = 0; i < BATCH_SIZE; ++i) {
        CUDA_CHECK(hipMalloc(&d_A[i], N * N * sizeof(FpType)));
        int streamIdx = i % NUM_STREAMS;
        CUDA_CHECK(hipMemcpyAsync(d_A[i], h_A[i], N * N * sizeof(FpType), 
                                  hipMemcpyHostToDevice, streams[streamIdx]));
    }

    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipMemcpy(d_A_array, d_A.data(), BATCH_SIZE * sizeof(FpType*), hipMemcpyHostToDevice));

    // Perform batched LU factorization
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    CUBLAS_CHECK(hipblasSgetrfBatched(handle, N, d_A_array, N, d_pivots, d_info, BATCH_SIZE));

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "cuBLAS Kernel execution time: " << milliseconds << " milliseconds\n";


    // Copy results back
    for (int i = 0; i < BATCH_SIZE; ++i) {
        int streamIdx = i % NUM_STREAMS;
        CUDA_CHECK(hipMemcpyAsync(h_L[i], d_A[i], N * N * sizeof(FpType), 
                                  hipMemcpyDeviceToHost, streams[streamIdx]));
    }

    CUDA_CHECK(hipDeviceSynchronize());

    // Verify LU decomposition
    std::cout << "Verifying LU decomposition..." << std::endl;
    int correctDecomp = 0, incorrectDecomp = 0;
    constexpr FpType TOLERANCE = 1e-3f;

    #pragma omp parallel for reduction(+:correctDecomp) schedule(dynamic)
    for (int b = 0; b < BATCH_SIZE; ++b) {
        // Extract L and U from the result
        for (int i = 0; i < N; ++i) {
            for (int j = 0; j < N; ++j) {
                if (i > j) {
                    h_L[b][i * N + j] = h_A[b][i * N + j];
                    h_U[b][i * N + j] = 0.0f;
                } else if (i == j) {
                    h_L[b][i * N + j] = 1.0f;
                    h_U[b][i * N + j] = h_A[b][i * N + j];
                } else {
                    h_L[b][i * N + j] = 0.0f;
                    h_U[b][i * N + j] = h_A[b][i * N + j];
                }
            }
        }

        // Verify L*U = A
        bool isCorrect = true;
        std::vector<FpType> result(N * N);
        
        #pragma omp simd collapse(2)
        for (int i = 0; i < N; ++i) {
            for (int j = 0; j < N; ++j) {
                FpType sum = 0.0f;
                for (int k = 0; k < N; ++k) {
                    sum += h_L[b][i * N + k] * h_U[b][k * N + j];
                }
                result[i * N + j] = sum;
                if (std::abs(result[i * N + j] - templateMatrix[i * N + j]) >= TOLERANCE) {
                    isCorrect = false;
                }
            }
        }
        
        if (isCorrect) correctDecomp++;
        else (!isCorrect) incorrectDecomp++;
    }

    std::cout << "Correct inversions: " << correctDecomp << << std::endl;
    std::cout << "Incorrect inversions: " << incorrectDecomp << std::endl;

    // Cleanup
    CUBLAS_CHECK(hipblasDestroy(handle));
    for (int i = 0; i < NUM_STREAMS; ++i) {
        CUDA_CHECK(hipStreamDestroy(streams[i]));
    }

    CUDA_CHECK(hipFree(d_A_array));
    CUDA_CHECK(hipFree(d_pivots));
    CUDA_CHECK(hipFree(d_info));

    for (int i = 0; i < BATCH_SIZE; ++i) {
        CUDA_CHECK(hipHostFree(h_A[i]));
        CUDA_CHECK(hipHostFree(h_L[i]));
        CUDA_CHECK(hipHostFree(h_U[i]));
        CUDA_CHECK(hipFree(d_A[i]));
    }

    return 0;
}